#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstring>
#include <cstdlib>
#include <iostream>
#include <vtkDoubleArray.h>
#include <vtkFloatArray.h>

#include "gmm_vtk_data_array.h"
#include <core/ndarray.h>
#include <core/thrust_common.h>
using namespace std;
namespace edda{
using namespace dist;


GmmVtkDataArray::~GmmVtkDataArray() { }

size_t GmmVtkDataArray::getLength() { return length; }

int GmmVtkDataArray::getNumComponents() { return this->components; }

GmmVtkDataArray::GmmVtkDataArray(vtkFieldData *fieldData, const char *arrayNamePrefix)  {
  target_comp = 0;
  char meanArrayName[1024];
  char stdevArrayName[1024];
  char varArrayName[1024];
  char weightArrayName[1024];
  for (int i=0; i<fieldData->GetNumberOfArrays(); i++)
  {
    sprintf(meanArrayName, "%smean%d", arrayNamePrefix, i );
    sprintf(stdevArrayName, "%sstdev%d", arrayNamePrefix, i );
    sprintf(varArrayName, "%svar%d", arrayNamePrefix, i );
    sprintf(weightArrayName, "%sweight%d", arrayNamePrefix, i );

    vtkSmartPointer<vtkDataArray> meanArray = fieldData->GetArray(meanArrayName);
    vtkSmartPointer<vtkDataArray> stdevArray = fieldData->GetArray(stdevArrayName);
    vtkSmartPointer<vtkDataArray> varArray = fieldData->GetArray(varArrayName);
    vtkSmartPointer<vtkDataArray> weightArray = fieldData->GetArray(weightArrayName);
    if (i==0 && meanArray && weightArray==0) {
      int c = meanArray->GetNumberOfComponents();
      int n = meanArray->GetNumberOfTuples();
      // allows when only one mean and one variance are provided
      // create weight Array
      weightArray = vtkSmartPointer<vtkFloatArray>::New();
      weightArray->SetNumberOfComponents(c);
      weightArray->SetNumberOfTuples(n);
      // asign 1's
      float *p = (float *)weightArray->GetVoidPointer(0);

      for (int j=0; j<n*c; j++)
        p[j] = 1.f;
    }
    if (meanArray && varArray && weightArray) {
      // get components
      int c = meanArray->GetNumberOfComponents();
      if (c != varArray->GetNumberOfComponents() || c!= weightArray->GetNumberOfComponents()) {
        printf("Warning: the number of array components do not match\n") ;
        c = 1;
      }
      this->components = c;

      // set arrays
      arrays.push_back(meanArray);
      arrays.push_back(varArray);
      arrays.push_back(weightArray);

    } else if (meanArray && stdevArray && weightArray) {
      // get components
      int c = meanArray->GetNumberOfComponents();
      if (c != stdevArray->GetNumberOfComponents() || c!= weightArray->GetNumberOfComponents()) {
        printf("Warning: the number of array components do not match\n") ;
        c = 1;
      }
      this->components = c;

      // convert stdev to variance
      arrays.push_back(meanArray);
      vtkSmartPointer<vtkFloatArray> varArray = vtkSmartPointer<vtkFloatArray>::New();
      int n = stdevArray->GetNumberOfTuples();
      varArray->SetNumberOfComponents(1);
      varArray->SetNumberOfTuples(n);
      for (int j=0; j<n; j++)
        *(float *)varArray->GetVoidPointer(j) = pow(stdevArray->GetTuple1(j), 2.) ;

      // set arrays
      arrays.push_back(varArray);
      arrays.push_back(weightArray);
    }
  }
  if (arrays.size() == 0) {
    length = 0;
    return;
  }
  length = arrays[0]->GetNumberOfTuples();
  for (size_t i=1; i<arrays.size(); i++)
  {
    length = std::min(length, (size_t)arrays[i]->GetNumberOfTuples());
  }
}

GmmVtkDataArray::GmmVtkDataArray(std::vector<vtkSmartPointer<vtkDataArray> > arrays_) {
  target_comp = 0;
  if (arrays_.size() == 0) {
    printf("Warning: no array assigned to GmmVtkArray\n");
    return;
  }
  if (arrays_.size() % 3 != 0) {
    printf("Warning: GmmVtkArray: some input arrays are truncated\n");
  }
  for (size_t i=0; i<arrays_.size()/3; i++) {
    this->arrays.push_back(arrays_[i]);
    this->arrays.push_back(arrays_[i+1]);
    this->arrays.push_back(arrays_[i+2]);
  }

  length = this->arrays[0]->GetNumberOfTuples();
  for (size_t i=0; i<this->arrays.size(); i++)
  {
    length = min(length, (size_t)this->arrays[i]->GetNumberOfTuples());
  }
}

dist::Variant GmmVtkDataArray::getDistr(size_t idx) {
  std::vector<GMMTuple> models ( arrays.size()/3 );
  for (size_t i=0; i<arrays.size(); i++) {
    models[i/3].p[i%3] = arrays[i]->GetComponent(idx, 0);
  }
  return DefaultGaussianMixture(models);
}

std::vector<dist::Variant> GmmVtkDataArray::getDistrVector(size_t idx) {
  int components = this->getNumComponents();
  std::vector<dist::Variant> v( components );
  for (int c = 0; c < components; c++ )
  {
    std::vector<GMMTuple> models(arrays.size()/3) ;
    for (size_t i=0; i<arrays.size(); i++) {
      models[i/3].p[i%3] = arrays[i]->GetComponent(idx, c);
    }
    v[c] = DefaultGaussianMixture(models);
  }
  return v;
}


Real GmmVtkDataArray::getScalar(size_t idx) {
  return getSample(getDistr(idx));
}

std::vector<Real> GmmVtkDataArray::getVector(size_t idx) {
  int components = this->getNumComponents();
  std::vector<Real> v( components );
  for (int c = 0; c < components; c++ )
  {
    std::vector<GMMTuple> models(arrays.size()/3) ;
    for (size_t i=0; i<arrays.size(); i++) {
      models[i/3].p[i%3] = arrays[i]->GetComponent(idx, c);
    }
    v[c] = getSample(DefaultGaussianMixture(models));
  }
  return v;
}



std::shared_ptr<GmmArray> GmmVtkDataArray::genNdArray() {
  //int n= arrays[i]->GetNumberOfTuples();
  //NdArray ndarray({length, arrays.size()});
  std::vector<NdArray<Real> > data(arrays.size());
  int n = length; // array length

  for (size_t i=0; i<arrays.size(); i++) {

    vtkFloatArray *farray = vtkFloatArray::SafeDownCast( arrays[i].Get() );
    vtkDoubleArray *darray = vtkDoubleArray::SafeDownCast( arrays[i].Get() );

    // Sometimes Real type that edda is using can be different from vtk array type
    // The following automatically converts vtk Float/Double Array to Real type

    if ((farray && sizeof(float) == sizeof(Real))   // alternative: typeid(Real) == typeid(float)
        || (darray && sizeof(double)==sizeof(Real)) ) {
      NdArray<Real> ndarray((Real *)arrays[i]->GetVoidPointer(0), {n} );

      data[i].take(ndarray);
    } else if ( sizeof(float) == sizeof(Real) ){
      // create a temp array in float
      vtkFloatArray *newArray = vtkFloatArray::New();
      newArray->DeepCopy(arrays[i]);
      NdArray<Real> ndarray((Real *)newArray->GetVoidPointer(0), {n} );
      data[i].take(ndarray);
      newArray->Delete();

    } else if ( sizeof(double) == sizeof(Real) ) {
      // create a temp array in double
      vtkDoubleArray *newArray = vtkDoubleArray::New();
      newArray->DeepCopy(arrays[i]);
      NdArray<Real> ndarray((Real *)newArray->GetVoidPointer(0), {n} );
      data[i].take(ndarray);
      newArray->Delete();

    } else {
      throw std::runtime_error("Real type not float or double.");
    }
  }
  return std::shared_ptr<GmmArray> ( new GmmArray(data) );

}
}; //edda
