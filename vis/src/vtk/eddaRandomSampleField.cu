// cuda related codes only
#include <iostream>

#include "eddaRandomSampleField.h"

#include "common.h"
#include "vtk_common.h"
#include "vtkInformation.h"
#include "vtkInformationVector.h"
#include "vtkObjectFactory.h"
#include "vtkStreamingDemandDrivenPipeline.h"

#include "io/gmm_vtk_data_array.h"
#include "core/thrust_random_sample.h"

using namespace std;
using namespace edda;


void eddaRandomSampleField::SampleDataArray(shared_ptr<GmmVtkDataArray> dataArray, vtkSmartPointer<vtkFieldData> output_field)
{
  // create output array
  vsp_new(vtkFloatArray, out_vtkArray);
  out_vtkArray->SetNumberOfComponents(dataArray->getNumComponents());
  out_vtkArray->SetNumberOfTuples(dataArray->getLength());
  out_vtkArray->SetName(ResultName.c_str());

#if 1 // thrust
  shared_ptr<GmmArray> gmmArray = dataArray->genNdArray();
  thrust::device_vector<Real> out(dataArray->getLength());

  randomSampleField(gmmArray->begin(), gmmArray->end(), out.begin());

  thrust::copy(out.begin(), out.end(), (float *)out_vtkArray->GetVoidPointer(0));

#else // sequential
  float *p = (float *)out_vtkArray->GetVoidPointer(0);
  int nc = dataArray->getNumComponents();
#pragma omp parallel for
  for (size_t i=0; i<dataArray->getLength(); i++)
  {
    if (nc==1) {
      p[i*nc] = dist::getSample( dataArray->getScalar(i) );
    } else {
      std::vector< dist::Variant > v = dataArray->getVector(i);
      for (int c=0; c<nc; c++)
        p[i*nc+c] = dist::getSample( v[c] );
    }
  }
#endif

  output_field->AddArray(out_vtkArray);
}
