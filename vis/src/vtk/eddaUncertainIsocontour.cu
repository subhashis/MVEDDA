// cuda related codes only
#include <iostream>

#include "eddaUncertainIsocontour.h"

#include "vtk_common.h"
#include "vtkCell.h"
#include "vtkCellData.h"
#include "vtkCharArray.h"
#include "vtkIdTypeArray.h"
#include "vtkImageData.h"
#include "vtkInformation.h"
#include "vtkInformationVector.h"
#include "vtkMath.h"
#include "vtkObjectFactory.h"
#include "vtkPointData.h"
#include "vtkStreamingDemandDrivenPipeline.h"

#include "io/gmm_vtk_data_array.h"
#include <filters/level_crossing_prob.h>

using namespace std;
using namespace edda;


void eddaUncertainIsocontour::Compute(vtkDataSet* input, int *dim,
  vtkDataSet* output)
{
  // process point data
  shared_ptr<GmmVtkDataArray> dataArray(new GmmVtkDataArray(input->GetPointData()) );
  shared_ptr<GmmArray> gmmArray = dataArray->genNdArray();

  // has point data?
  if (dataArray->getLength() > 0) {
    int out_length = (dim[0]-1)*(dim[1]-1)*(dim[2]-1);

    shared_ptr<NdArray<float> > out_ndarray;

    //ReturnStatus r = levelCrossingSerial(dataArray.get(), dim, this->Isov, (float *)out_vtkArray->GetVoidPointer(0));
    ReturnStatus r = levelCrossingProb(gmmArray->begin(), dim, this->Isov, out_ndarray);

    if (r!=ReturnStatus::SUCCESS) {
      return ;
    }

    // create output array
    vsp_new(vtkFloatArray, out_vtkArray);
    out_vtkArray->SetNumberOfComponents(1);
    out_vtkArray->SetNumberOfTuples(out_length);
    out_vtkArray->SetName("ProbField");
    // copy from device to host
    out_ndarray->copy_to_host((float *)out_vtkArray->GetVoidPointer(0));

    output->GetCellData()->AddArray(out_vtkArray);
  } else {
    // has cell data?
    //dataArray = shared_ptr<GmmVtkDataArray>(new GmmVtkDataArray(input->GetCellData()) );
  }

}
